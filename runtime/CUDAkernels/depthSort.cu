#include "hip/hip_runtime.h"
#include "bonsai.h"
#include <thrust/device_ptr.h>
#include <thrust/sort.h>
#include "vector_math.h"
#include <cassert>

#ifdef USE_OPENGL
  #include <GL/glew.h>
  #include <cuda_gl_interop.h>
#endif

// calculate eye-space depth for each particle
KERNEL_DECLARE(calcDepthKernel)(float4 *pos, float *depth, int *indices, float4 modelViewZ, int numParticles)
{
    int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i >= numParticles) return;

	float4 p = pos[i];
	float z = dot(make_float4(p.x, p.y, p.z, 1.0f), modelViewZ);
	
	depth[i] = z;
	indices[i] = i;
}

void thrustSort(float* keys, int* values, int count)
{
    thrust::device_ptr<float> dkeys(keys);
    thrust::device_ptr<int> dvalues(values);
    thrust::sort_by_key(dkeys, dkeys + count, dvalues);
}

extern "C"
void initCUDA()
{
  #ifdef USE_OPENGL
    cudaGLSetGLDevice(0);
  #endif
}

extern "C"
void depthSortCUDA(float4 *pos, float *depth, int *indices, float4 modelViewZ, int numParticles)
{
	int numThreads = 256;
	int numBlocks = (numParticles + numThreads - 1) / numThreads;
    calcDepthKernel<<< numBlocks, numThreads >>>(pos, depth, indices, modelViewZ, numParticles);

	thrustSort(depth, indices, numParticles);
}

// integer hash function (credit: rgba/iq)
  __device__
  int ihash(int n)
  {
      n=(n<<13)^n;
      return (n*(n*n*15731+789221)+1376312589) & 0x7fffffff;
  }

  // returns random float between 0 and 1
  __device__
  float frand(int n)
  {
	  return ihash(n) / 2147483647.0f;
  }

#if 0  /* Simon's code */
__global__
void assignColorsKernel(float4 *colors, int *ids, int numParticles, 
	float4 color2, float4 color3, float4 color4, 
	float4 starColor, float4 bulgeColor, float4 darkMatterColor, float4 dustColor,
	int m_brightFreq)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if( tid >= numParticles ) return;

	int id =  ids[tid];

	float r = frand(id);
	//float4 color = { r, 1-r, 0.5f, 1.0f };
	//float4 color = { 1.0f, 0.0f, 0.0f, 1.0f };

	float4 color;

      if (id >= 0 && id < 40000000)     //Disk
      {
        color = ((id % m_brightFreq) != 0) ? 
        starColor :
        ((id / m_brightFreq) & 1) ? color2 : color3;
      } else if (id >= 40000000 && id < 50000000)     // Glowing stars in spiral arms
      {
        color = ((id%4) == 0) ? color4 : color3;
      }
				else if (id >= 50000000 && id < 70000000) //Dust
				{
					color = dustColor * make_float4(r, r, r, 1.0f);
				} 
				else if (id >= 70000000 && id < 100000000) // Glow massless dust particles
				{
					color = color3;  /*  adds glow in purple */
				}
      else if (id >= 100000000 && id < 200000000) //Bulge
      {
		  //colors[i] = starColor;
        color = bulgeColor;
	  } 
      else //>= 200000000, Dark matter
      {
        color = darkMatterColor;
		  //colors[i] = darkMatterColor * make_float4(r, r, r, 1.0f);
      }            
      
  
	colors[tid] = color;
}
#else  /* Ev's code :) */

class StarSampler
{
	private:
		float slope;
		float slope1;
		float slope1inv;
		float Mu_lo;
		float C;
		int   N;
		float *Masses;
		float4 *Colours;

	public:

		__device__ StarSampler(
				const float _N, 
				float  *_Masses, 
				float4 *_Colours,
				const float _slope = -2.35) : 
			slope(_slope), N(_N), Masses(_Masses), Colours(_Colours)
	{
		const float Mhi = Masses[0];
		const float Mlo = Masses[N-1];
		slope1    = slope + 1.0f;
//		assert(slope1 != 0.0f);
	  slope1inv	= 1.0f/slope1;

		Mu_lo = __powf(Mlo, slope1);
		C = (powf(Mhi, slope1) - powf(Mlo, slope1));
	}

		__device__ float sampleMass(const int id)  const
		{
			const float Mu = C*frand(id) + Mu_lo;
//			assert(Mu > 0.0);
			const float M   = __powf(Mu, slope1inv);
			return M;
		}

		__device__ float4 getColour(const float M) const
		{
			int beg = 0;
			int end = N;
			int mid = (beg + end) >> 1;
			while (end - beg > 1)
			{
				if (Masses[mid] > M)
					beg = mid;
				else 
					end = mid;
				mid = (beg + end) >> 1;
			}

			return Colours[mid];
		}
};

KERNEL_DECLARE(assignColorsKernel) (float4 *colors, int *ids, float4 *col, int numParticles, 
		float4 color2, float4 color3, float4 color4, 
		float4 starColor, float4 bulgeColor, float4 darkMatterColor, float4 dustColor,
		int m_brightFreq, float4 t_current)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if( tid >= numParticles ) return;

	int id =  ids[tid];
	
	float r = frand(id);
	float4 color;
	
        // setup //
	if(id > 100000000 && id < 200000000){
		//every 3000th particle is lit up
		//shiny particles for all players!
		color = (((id%3000) == 0) || ((id%3000) == 1) || ((id%3000) == 2) || ((id%3000) == 3)) ? make_float4(col[tid].x+0.3,col[tid].y+0.3,col[tid].z+0.3,col[tid].w*25.0) : col[tid];
	}
	else{
	  color = col[tid];
	}
	colors[tid] = color;	

}
#endif

	extern "C"
void assignColors(float4 *colors, int *ids, float4 *col, int numParticles, //int *col,int *ids, , float4 *col //, real4 *col
		float4 color2, float4 color3, float4 color4, 
		float4 starColor, float4 bulgeColor, float4 darkMatterColor, float4 dustColor,
		int m_brightFreq, float4  t_current)
{
	int numThreads = 256;
// 	printf("number of particles %d\n",numParticles);
	
	int numBlocks = (numParticles + numThreads - 1) / numThreads;
	assignColorsKernel<<< numBlocks, numThreads >>>(colors, ids, col, numParticles, 
			color2, color3, color4, starColor, bulgeColor, darkMatterColor, dustColor, m_brightFreq, t_current);
}
